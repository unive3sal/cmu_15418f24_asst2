#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"


extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2.
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void upsweep_kernel(int* data, int N, int twod, int twod1) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < N && i % twod1 == 0) {
        data[i + twod1 - 1] += data[i + twod - 1];
    }
}

__global__ void downsweep_kernel(int* data, int N, int twod, int twod1) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N && i % twod1 == 0) {        int t = data[i + twod - 1];
        data[i + twod - 1] = data[i + twod1 - 1];
        data[i + twod1 - 1] += t;
    }
}

void exclusive_scan(int* device_data, int length)
{
    /* TODO
     * Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the data in device memory
     * The data are initialized to the inputs.  Your code should
     * do an in-place scan, generating the results in the same array.
     * This is host code -- you will need to declare one or more CUDA
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the data array is sized to accommodate the next
     * power of 2 larger than the input.
     */
    const int threads_per_block = 512;
    const int blocks = (length + threads_per_block - 1) / threads_per_block;

    const int N = nextPow2(length);
    for (int twod = 1; twod < length; twod *= 2) {
        int twod1 = 2 * twod;
        upsweep_kernel<<<blocks, threads_per_block>>>(device_data, N, twod, twod1);
        hipDeviceSynchronize();
    }
    hipMemset(device_data + length - 1, 0, 1);

    for (int twod = length / 2; twod >= 1; twod /= 2) {
        int twod1 = 2 * twod;
        downsweep_kernel<<<blocks, threads_per_block>>>(device_data, N, twod, twod1);
        hipDeviceSynchronize();
    }
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness.
    // You may have an easier time in your implementation if you assume the
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;

    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}

__global__ void find_peaks_kernel(bool* out, int* prefix_sum, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && i < len - 1) {
        int current_num = prefix_sum[i + 1] - prefix_sum[i];
        int preceed_num = prefix_sum[i] - prefix_sum[i - 1];
        int follow_num = prefix_sum[i + 2] - prefix_sum[i + 1];
        if (current_num > preceed_num && current_num > follow_num) {
            out[i] = true;
        }
    }
}

int find_peaks(int *device_input, int length, int *device_output) {
    /* TODO:
     * Finds all elements in the list that are greater than the elements before and after,
     * storing the index of the element into device_result.
     * Returns the number of peak elements found.
     * By definition, neither element 0 nor element length-1 is a peak.
     *
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if
     * it requires that. However, you must ensure that the results of
     * find_peaks are correct given the original length.
     */
    const int threads_per_block = 512;
    const int blocks = (length + threads_per_block - 1) / threads_per_block;

    // a[i] = prefix[i + 1] - prefix[i]
    // peak: a[i] > a[i - 1] && a[i] > a[i + 1]
    int last = device_input[length - 1];
    exclusive_scan(device_input, length);   // raw data -> prefix sum

    int* prefix_sum;
    bool* flag_map;
    hipMalloc(&prefix_sum, (length + 1) * sizeof(int));
    hipMalloc(&flag_map, length * sizeof(bool));
    hipMemcpy(prefix_sum, device_input, length * sizeof(int), hipMemcpyDeviceToDevice);
    //hipMemset(prefix_sum + length, last + prefix_sum[length - 1], 1);
    find_peaks_kernel<<<blocks, threads_per_block>>>(flag_map, prefix_sum, length);
    hipDeviceSynchronize();

    int out_idx = 0;
    /*
    for (int i = 0; i < length; ++i) {
        if (flag_map[i]) {
            device_output[out_idx++] = i;
        }
    }
    */

    hipFree(prefix_sum);
    hipFree(flag_map);

    return out_idx;
}



/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    int result = find_peaks(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}


void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
